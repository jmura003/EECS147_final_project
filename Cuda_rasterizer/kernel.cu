typedef unsigned char Rgb[3]; 
typedef float Vec2[2]; 
typedef float Vec3[3]; 


#include <hip/hip_runtime.h>
#include <stdio.h>

#define ww 16
#define hh 16

__device__ inline 
float edgeFunction(const Vec2 &a, const Vec2 &b, const Vec2 &c) 
{ return (c[0] - a[0]) * (b[1] - a[1]) - (c[1] - a[1]) * (b[0] - a[0]); } 

__global__ void rasterize_triangle(unsigned char * framebuffer_d, const float * v0_d, const float * v1_d, const float * v2_d, 
    const int w, const int h){

        
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int j = ty + blockIdx.y * blockDim.y; // rows
        int i = tx + blockIdx.x * blockDim.x; // cols

 

        Vec2 a = {v0_d[0], v0_d[1]};
        Vec2 b = {v1_d[0], v1_d[1]};
        Vec2 c = {v2_d[0], v2_d[1]};

        float area = edgeFunction(a,b,c);
        //for(int j = threadIdx.y; j < w; j += blockDim.y){
        //    for(int i = threadIdx.x; i < h; i += blockDim.x){
        //for(int idx = 0; idx < w; idx++){
        Vec2 p = {i + 0.5f, j + 0.5f};
        int index = (i + j * w)*3;   
        float alpha = edgeFunction(b,c,p);
        float beta = edgeFunction(c,a,p);
        float gamma = edgeFunction(a,b,p);
        if(alpha >= 0 && beta >= 0 && gamma >= 0){
            alpha = alpha / area;
            beta = beta / area;
            gamma = gamma / area;
            float r = alpha;
            float g = beta;
            float bb = gamma;
            if(i < 512 && j < 512){
                framebuffer_d[index] = (unsigned char)(r * 255);
                framebuffer_d[index + 1] = (unsigned char)(g * 255);
                framebuffer_d[index + 2] = (unsigned char)(bb * 255);
            }
        }
            //}
            //}
        //}

}

void basicTriRast(unsigned char * framebuffer_d, const float * v0_d, const float * v1_d, const float * v2_d, const int w, const int h, const int SIZE){
    //dim3 DimGrid(ceil((w*h)/512.0),1,1);
    const unsigned int BLOCK_SIZE = 32;
    //ceil(double(512)/BLOCK_SIZE)
    dim3 BlocksPerGrid(ceil(double(512)/BLOCK_SIZE),ceil(double(512)/BLOCK_SIZE),1);
    dim3 ThreadsPerBlock(BLOCK_SIZE,BLOCK_SIZE,1);
    // dim3 ThreadsPerBlock(256,1,1);
    // dim3 BlocksPerGrid(ceil(double(SIZE)/BLOCK_SIZE),1,1);
    rasterize_triangle<<<BlocksPerGrid, ThreadsPerBlock>>>(framebuffer_d, v0_d, v1_d, v2_d, w, h);
}